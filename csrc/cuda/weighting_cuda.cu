#include "hip/hip_runtime.h"
#include "weighting_cuda.h"

#include <ATen/cuda/HIPContext.h>

#include "atomics.cuh"
#include "utils.cuh"

#define THREADS 1024
#define BLOCKS(N) (N + THREADS - 1) / THREADS

template <typename scalar_t>
__global__ void
spline_weighting_fw_kernel(const scalar_t *x, const scalar_t *weight,
                           const scalar_t *basis, const int64_t *weight_index,
                           scalar_t *out, int64_t E, int64_t M_in,
                           int64_t M_out, int64_t S, int64_t numel) {

  const int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int64_t e = thread_idx / M_out;
  const int64_t m_out = thread_idx % M_out;

  if (thread_idx < numel) {
    scalar_t v = (scalar_t)0.;

    for (ptrdiff_t s = 0; s < S; s++) {
      const scalar_t b = basis[e * S + s];
      const int64_t wi = weight_index[e * S + s];
      for (int64_t m_in = 0; m_in < M_in; m_in++) {
        scalar_t tmp = weight[wi * M_in * M_out + m_in * M_out + m_out];
        tmp *= b * x[e * M_in + m_in];
        v += tmp;
      }
    }
    out[thread_idx] = v;
  }
}

torch::Tensor spline_weighting_fw_cuda(torch::Tensor x, torch::Tensor weight,
                                       torch::Tensor basis,
                                       torch::Tensor weight_index) {
  CHECK_CUDA(x);
  CHECK_CUDA(weight);
  CHECK_CUDA(basis);
  CHECK_CUDA(weight_index);
  c10::cuda::MaybeSetDevice(x.get_device());

  CHECK_INPUT(x.size(1) == weight.size(1));

  auto E = x.size(0);
  auto M_in = x.size(1);
  auto M_out = weight.size(2);
  auto S = basis.size(1);

  auto out = at::empty({E, M_out}, x.options());

  auto weight_index_data = weight_index.data_ptr<int64_t>();

  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "weighting_fw", [&] {
    auto x_data = x.data_ptr<scalar_t>();
    auto weight_data = weight.data_ptr<scalar_t>();
    auto basis_data = basis.data_ptr<scalar_t>();
    auto out_data = out.data_ptr<scalar_t>();

    spline_weighting_fw_kernel<scalar_t>
        <<<BLOCKS(out.numel()), THREADS, 0, stream>>>(
            x_data, weight_data, basis_data, weight_index_data, out_data, E,
            M_in, M_out, S, out.numel());
  });

  return out;
}

template <typename scalar_t>
__global__ void
spline_weighting_bw_x_kernel(const scalar_t *grad_out, const scalar_t *weight,
                             const scalar_t *basis, const int64_t *weight_index,
                             scalar_t *grad_x, int64_t E, int64_t M_in,
                             int64_t M_out, int64_t S, int64_t numel) {

  const int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int64_t e = thread_idx / M_in;
  const int64_t m_in = thread_idx % M_in;

  if (thread_idx < numel) {
    scalar_t v = (scalar_t)0.;

    for (int64_t s = 0; s < S; s++) {
      const scalar_t b = basis[e * S + s];
      const int64_t wi = weight_index[e * S + s];

      for (int64_t m_out = 0; m_out < M_out; m_out++) {
        scalar_t tmp = weight[wi * M_out * M_in + m_out * M_in + m_in];
        tmp *= b * grad_out[e * M_out + m_out];
        v += tmp;
      }
    }
    grad_x[thread_idx] = v;
  }
}

torch::Tensor spline_weighting_bw_x_cuda(torch::Tensor grad_out,
                                         torch::Tensor weight,
                                         torch::Tensor basis,
                                         torch::Tensor weight_index) {
  CHECK_CUDA(grad_out);
  CHECK_CUDA(weight);
  CHECK_CUDA(basis);
  CHECK_CUDA(weight_index);
  c10::cuda::MaybeSetDevice(grad_out.get_device());

  CHECK_INPUT(grad_out.size(1) == weight.size(2));

  auto E = grad_out.size(0);
  auto M_in = weight.size(1);
  auto M_out = grad_out.size(1);
  auto S = basis.size(1);

  auto grad_x = at::zeros({E, M_in}, grad_out.options());
  weight = weight.transpose(1, 2).contiguous(); // Contiguous memory-access.

  auto weight_index_data = weight_index.data_ptr<int64_t>();

  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES(grad_out.scalar_type(), "weighting_bw_x", [&] {
    auto grad_out_data = grad_out.data_ptr<scalar_t>();
    auto weight_data = weight.data_ptr<scalar_t>();
    auto basis_data = basis.data_ptr<scalar_t>();
    auto grad_x_data = grad_x.data_ptr<scalar_t>();

    spline_weighting_bw_x_kernel<scalar_t>
        <<<BLOCKS(grad_x.numel()), THREADS, 0, stream>>>(
            grad_out_data, weight_data, basis_data, weight_index_data,
            grad_x_data, E, M_in, M_out, S, grad_x.numel());
  });

  return grad_x;
}

template <typename scalar_t>
__global__ void spline_weighting_bw_weight_kernel(
    const scalar_t *grad_out, const scalar_t *x, const scalar_t *basis,
    const int64_t *weight_index, scalar_t *grad_weight, int64_t E, int64_t M_in,
    int64_t M_out, int64_t S, int64_t numel) {

  const int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int64_t e = thread_idx / M_out;
  const int64_t m_out = thread_idx % M_out;

  if (thread_idx < numel) {
    auto g = grad_out[e * M_out + m_out];
    for (int64_t s = 0; s < S; s++) {
      const scalar_t b = basis[e * S + s];
      const int64_t wi = weight_index[e * S + s];

      for (int64_t m_in = 0; m_in < M_in; m_in++) {
        auto v = g * b * x[e * M_in + m_in];
        atomAdd(&grad_weight[wi * M_in * M_out + m_in * M_out + m_out], v);
      }
    }
  }
}

torch::Tensor spline_weighting_bw_weight_cuda(torch::Tensor grad_out,
                                              torch::Tensor x,
                                              torch::Tensor basis,
                                              torch::Tensor weight_index,
                                              int64_t kernel_size) {
  CHECK_CUDA(grad_out);
  CHECK_CUDA(x);
  CHECK_CUDA(basis);
  CHECK_CUDA(weight_index);
  c10::cuda::MaybeSetDevice(grad_out.get_device());

  auto E = grad_out.size(0);
  auto M_in = x.size(1);
  auto M_out = grad_out.size(1);
  auto S = basis.size(1);

  auto grad_weight = at::zeros({kernel_size, M_in, M_out}, grad_out.options());

  auto weight_index_data = weight_index.data_ptr<int64_t>();

  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "weighting_bw_weight", [&] {
    auto grad_out_data = grad_out.data_ptr<scalar_t>();
    auto x_data = x.data_ptr<scalar_t>();
    auto basis_data = basis.data_ptr<scalar_t>();
    auto grad_weight_data = grad_weight.data_ptr<scalar_t>();

    spline_weighting_bw_weight_kernel<scalar_t>
        <<<BLOCKS(grad_out.numel()), THREADS, 0, stream>>>(
            grad_out_data, x_data, basis_data, weight_index_data,
            grad_weight_data, E, M_in, M_out, S, grad_out.numel());
  });

  return grad_weight;
}

template <typename scalar_t>
__global__ void spline_weighting_bw_basis_kernel(
    const scalar_t *grad_out, const scalar_t *x, const scalar_t *weight,
    const int64_t *weight_index, scalar_t *grad_basis, int64_t E, int64_t M_in,
    int64_t M_out, int64_t S, int64_t numel) {

  const size_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int64_t e = thread_idx / M_out;
  const int64_t m_out = thread_idx % M_out;

  if (thread_idx < numel) {
    const scalar_t g = grad_out[e * M_out + m_out];

    for (int64_t s = 0; s < S; s++) {
      scalar_t v = (scalar_t)0.;
      const int64_t wi = weight_index[e * S + s];

      for (int64_t m_in = 0; m_in < M_in; m_in++) {
        const scalar_t w = weight[wi * M_in * M_out + m_in * M_out + m_out];
        v += g * w * x[e * M_in + m_in];
      }
      atomAdd(&grad_basis[e * S + s], v);
    }
  }
}

torch::Tensor spline_weighting_bw_basis_cuda(torch::Tensor grad_out,
                                             torch::Tensor x,
                                             torch::Tensor weight,
                                             torch::Tensor weight_index) {
  CHECK_CUDA(grad_out);
  CHECK_CUDA(x);
  CHECK_CUDA(weight);
  CHECK_CUDA(weight_index);
  c10::cuda::MaybeSetDevice(grad_out.get_device());

  CHECK_INPUT(x.size(1) == weight.size(1));
  CHECK_INPUT(grad_out.size(1) == weight.size(2));

  auto E = grad_out.size(0);
  auto M_in = x.size(1);
  auto M_out = grad_out.size(1);
  auto S = weight_index.size(1);

  auto grad_basis = at::zeros({E, S}, grad_out.options());

  auto weight_index_data = weight_index.data_ptr<int64_t>();

  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "weighting_bw_basis", [&] {
    auto grad_out_data = grad_out.data_ptr<scalar_t>();
    auto x_data = x.data_ptr<scalar_t>();
    auto weight_data = weight.data_ptr<scalar_t>();
    auto grad_basis_data = grad_basis.data_ptr<scalar_t>();

    spline_weighting_bw_basis_kernel<scalar_t>
        <<<BLOCKS(grad_out.numel()), THREADS, 0, stream>>>(
            grad_out_data, x_data, weight_data, weight_index_data,
            grad_basis_data, E, M_in, M_out, S, grad_out.numel());
  });

  return grad_basis;
}
