#include "hip/hip_runtime.h"
#include "basis_cuda.h"

#include <ATen/cuda/HIPContext.h>

#include "utils.cuh"

#define THREADS 1024
#define BLOCKS(N) (N + THREADS - 1) / THREADS

template <typename scalar_t, int64_t degree> struct Basis {
  static inline __device__ scalar_t forward(scalar_t v, int64_t k_mod) {
    if (degree == 1) {
      return 1. - v - k_mod + 2. * v * k_mod;
    } else if (degree == 2) {
      if (k_mod == 0)
        return 0.5 * v * v - v + 0.5;
      else if (k_mod == 1)
        return -v * v + v + 0.5;
      else
        return 0.5 * v * v;
    } else if (degree == 3) {
      if (k_mod == 0)
        return (1. - v) * (1. - v) * (1. - v) / 6.;
      else if (k_mod == 1)
        return (3. * v * v * v - 6. * v * v + 4.) / 6.;
      else if (k_mod == 2)
        return (-3. * v * v * v + 3. * v * v + 3. * v + 1.) / 6.;
      else
        return v * v * v / 6.;
    } else {
      return (scalar_t)-1.;
    }
  }

  static inline __device__ scalar_t backward(scalar_t v, int64_t k_mod) {
    if (degree == 1) {
      return 2 * k_mod - 1;
    } else if (degree == 2) {
      if (k_mod == 0)
        return v - 1.;
      else if (k_mod == 1)
        return -2. * v + 1.;
      else
        return v;
    } else if (degree == 3) {
      if (k_mod == 0)
        return (-v * v + 2. * v - 1.) / 2.;
      else if (k_mod == 1)
        return (3. * v * v - 4. * v) / 2.;
      else if (k_mod == 2)
        return (-3. * v * v + 2. * v + 1.) / 2.;
      else
        return v * v / 2.;
    } else {
      return (scalar_t)-1.;
    }
  }
};

template <typename scalar_t, int64_t degree>
__global__ void
spline_basis_fw_kernel(const scalar_t *pseudo, const int64_t *kernel_size,
                       const uint8_t *is_open_spline, scalar_t *basis,
                       int64_t *weight_index, int64_t E, int64_t D, int64_t S,
                       int64_t numel) {

  const int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int64_t e = thread_idx / S;
  const int64_t s = thread_idx % S;

  if (thread_idx < numel) {
    int64_t k = s, wi = 0, wi_offset = 1;
    scalar_t b = (scalar_t)1.;

    for (int64_t d = 0; d < D; d++) {
      const int64_t k_mod = k % (degree + 1);
      k /= degree + 1;

      scalar_t v = pseudo[e * D + d];
      v *= kernel_size[d] - degree * is_open_spline[d];

      wi += (((int64_t)v + k_mod) % kernel_size[d]) * wi_offset;
      wi_offset *= kernel_size[d];

      v -= floor(v);
      v = Basis<scalar_t, degree>::forward(v, k_mod);
      b *= v;
    }

    basis[thread_idx] = b;
    weight_index[thread_idx] = wi;
  }
}

std::tuple<torch::Tensor, torch::Tensor>
spline_basis_fw_cuda(torch::Tensor pseudo, torch::Tensor kernel_size,
                     torch::Tensor is_open_spline, int64_t degree) {
  CHECK_CUDA(pseudo);
  CHECK_CUDA(kernel_size);
  CHECK_CUDA(is_open_spline);
  c10::cuda::MaybeSetDevice(pseudo.get_device());

  CHECK_INPUT(kernel_size.dim() == 1);
  CHECK_INPUT(pseudo.size(1) == kernel_size.numel());
  CHECK_INPUT(is_open_spline.dim());
  CHECK_INPUT(pseudo.size(1) == is_open_spline.numel());

  auto E = pseudo.size(0);
  auto D = pseudo.size(1);
  auto S = (int64_t)(powf(degree + 1, D) + 0.5);

  auto basis = at::empty({E, S}, pseudo.options());
  auto weight_index = at::empty({E, S}, kernel_size.options());

  auto kernel_size_data = kernel_size.data_ptr<int64_t>();
  auto is_open_spline_data = is_open_spline.data_ptr<uint8_t>();
  auto weight_index_data = weight_index.data_ptr<int64_t>();

  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES(pseudo.scalar_type(), "basis_fw", [&] {
    auto pseudo_data = pseudo.data_ptr<scalar_t>();
    auto basis_data = basis.data_ptr<scalar_t>();

    AT_DISPATCH_DEGREE_TYPES(degree, [&] {
      spline_basis_fw_kernel<scalar_t, DEGREE>
          <<<BLOCKS(basis.numel()), THREADS, 0, stream>>>(
              pseudo_data, kernel_size_data, is_open_spline_data, basis_data,
              weight_index_data, E, D, S, basis.numel());
    });
  });

  return std::make_tuple(basis, weight_index);
}

template <typename scalar_t, int64_t degree>
__global__ void
spline_basis_bw_kernel(const scalar_t *grad_basis, const scalar_t *pseudo,
                       const int64_t *kernel_size,
                       const uint8_t *is_open_spline, scalar_t *grad_pseudo,
                       int64_t E, int64_t D, int64_t S, int64_t numel) {

  const int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int64_t e = thread_idx / D;
  const int64_t d = thread_idx % D;

  if (thread_idx < numel) {
    scalar_t g = (scalar_t)0., tmp;

    for (ptrdiff_t s = 0; s < S; s++) {
      int64_t k_mod = (s / (int64_t)(powf(degree + 1, d) + 0.5)) % (degree + 1);

      scalar_t v = pseudo[e * D + d];
      v *= kernel_size[d] - degree * is_open_spline[d];
      v -= floor(v);
      v = Basis<scalar_t, degree>::backward(v, k_mod);
      tmp = v;

      for (int64_t d_it = 1; d_it < D; d_it++) {
        const int64_t d_new = d_it - (d >= d_it);
        k_mod = (s / (int64_t)(powf(degree + 1, d_new) + 0.5)) % (degree + 1);
        v = pseudo[e * D + d_new];
        v *= kernel_size[d_new] - degree * is_open_spline[d_new];
        v -= floor(v);
        v = Basis<scalar_t, degree>::forward(v, k_mod);
        tmp *= v;
      }
      g += tmp * grad_basis[e * S + s];
    }
    g *= kernel_size[d] - degree * is_open_spline[d];
    grad_pseudo[thread_idx] = g;
  }
}

torch::Tensor spline_basis_bw_cuda(torch::Tensor grad_basis,
                                   torch::Tensor pseudo,
                                   torch::Tensor kernel_size,
                                   torch::Tensor is_open_spline,
                                   int64_t degree) {
  CHECK_CUDA(grad_basis);
  CHECK_CUDA(pseudo);
  CHECK_CUDA(kernel_size);
  CHECK_CUDA(is_open_spline);
  c10::cuda::MaybeSetDevice(grad_basis.get_device());

  CHECK_INPUT(grad_basis.size(0) == pseudo.size(0));
  CHECK_INPUT(kernel_size.dim() == 1);
  CHECK_INPUT(pseudo.size(1) == kernel_size.numel());
  CHECK_INPUT(is_open_spline.dim());
  CHECK_INPUT(pseudo.size(1) == is_open_spline.numel());

  auto E = pseudo.size(0);
  auto D = pseudo.size(1);
  auto S = grad_basis.size(1);

  auto grad_pseudo = at::empty({E, D}, pseudo.options());

  auto kernel_size_data = kernel_size.data_ptr<int64_t>();
  auto is_open_spline_data = is_open_spline.data_ptr<uint8_t>();

  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES(pseudo.scalar_type(), "basis_bw", [&] {
    auto grad_basis_data = grad_basis.data_ptr<scalar_t>();
    auto pseudo_data = pseudo.data_ptr<scalar_t>();
    auto grad_pseudo_data = grad_pseudo.data_ptr<scalar_t>();

    AT_DISPATCH_DEGREE_TYPES(degree, [&] {
      spline_basis_bw_kernel<scalar_t, DEGREE>
          <<<BLOCKS(grad_pseudo.numel()), THREADS, 0, stream>>>(
              grad_basis_data, pseudo_data, kernel_size_data,
              is_open_spline_data, grad_pseudo_data, E, D, S,
              grad_pseudo.numel());
    });
  });

  return grad_pseudo;
}
